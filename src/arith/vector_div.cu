#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "arith/arith.cuh"

// C = A / B
__global__ void vectorDivKernel(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float d = B[i];
        C[i] = (fabsf(d) > 1e-8f) ? (A[i] / d) : 0.0f;
    }
}

void vectorDiv(const float* A, const float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    // Allocate GPU memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    vectorDivKernel<<<blocks, threads>>>(d_A, d_B, d_C, N);

    hipDeviceSynchronize();

    // Copy result back
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
