#include "hip/hip_runtime.h"
/**
 * @file arith/vector_log.cu
 * @brief Elementwise natural log of a floats array on the GPU.
 */

#include <hip/hip_runtime.h>
#include "arith/arith.cuh"

/**
 * @brief Performs elementwise logarithm: B[i] = log(A[i])
 * @param A Pointer to input array A
 * @param B Pointer to output array B
 * @param N Number of elements
 */
__global__ void vectorLogKernel(const float* A, float* B, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        B[i] = logf(A[i]);
}

/**
 * @brief Host launcher for the elementwise natural logarithm kernel.
 * 
 * @param A Pointer to device array A.
 * @param B Pointer to device array for results.
 * @param N Number of elements to process.
 */
void vectorLog(const float* A, float* B, int N) {
    float *d_A, *d_B;
    size_t size = N * sizeof(float);

    // Allocate GPU memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);

    // Copy data to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    vectorLogKernel<<<blocks, threads>>>(d_A, d_B, N);

    hipDeviceSynchronize();

    // Copy result back
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
}
