#include "hip/hip_runtime.h"
#include "vector_add.cuh"
#include <hip/hip_runtime.h>

// C = A + B
__global__ void vectorAddKernel(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

void vectorAdd(const float* A, const float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    // Allocate GPU memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    vectorAddKernel<<<blocks, threads>>>(d_A, d_B, d_C, N);

    hipDeviceSynchronize();

    // Copy result back
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
