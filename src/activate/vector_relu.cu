#include "hip/hip_runtime.h"
/**
 * @file activate/vector_relu.cu
 * @brief Elementwise ReLU of a floats array on the GPU.
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include "activate/activate.cuh"
#define CHECK_CUDA(call) do {                                 \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
        std::cerr << "CUDA error at " << __FILE__ << ":"      \
                  << __LINE__ << " — "                        \
                  << hipGetErrorString(err) << std::endl;    \
        exit(EXIT_FAILURE);                                   \
    }                                                         \
} while (0)
/**
 * @brief Performs elementwise ReLU: B[i] = ReLU(A[i])
 * @param A Pointer to input array A
 * @param B Pointer to output array B
 * @param N Number of elements
 */
__global__ void vectorReLUKernel(const float* A, float* B, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        B[i] = fmaxf(0.0f, A[i]);
}

/**
 * @brief Host launcher for the elementwise ReLU kernel.
 * 
 * @param A Pointer to device array A.
 * @param B Pointer to device array for results.
 * @param N Number of elements to process.
 */
void vectorReLU(const float* A, float* B, int N) {
    float *d_A, *d_B;
    size_t size = N * sizeof(float);

    // Allocate GPU memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);

    // Copy data to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    vectorReLUKernel<<<blocks, threads>>>(d_A, d_B, N);
    CHECK_CUDA(hipGetLastError());           // catch launch errors

    hipDeviceSynchronize();

    // Copy result back
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
}
