#include "hip/hip_runtime.h"
/**
 * @file activate/vector_gelu.cu
 * @brief Elementwise GELU of a floats array on the GPU.
 */

#define _USE_MATH_DEFINES
#include <hip/hip_runtime.h>
#include <math.h>
#include "activate/activate.cuh"

/**
 * @brief Performs elementwise GELU: B[i] = GELU(A[i])
 * @param A Pointer to input array A
 * @param B Pointer to output array B
 * @param N Number of elements
 */
__global__ void vectorGELUKernel(const float* A, float* B, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float x = A[i];
        float inner = sqrtf(2.0f / M_PI) * (x + 0.044715f * x * x * x);
        float gelu = 0.5f * x * (1.0f + tanhf(inner));
        B[i] = gelu;
    }
}

/**
 * @brief Host launcher for the elementwise GELU kernel.
 * 
 * @param A Pointer to device array A.
 * @param B Pointer to device array for results.
 * @param N Number of elements to process.
 */
void vectorGELU(const float* A, float* B, int N) {
    float *d_A, *d_B;
    size_t size = N * sizeof(float);

    // Allocate GPU memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);

    // Copy data to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    vectorGELUKernel<<<blocks, threads>>>(d_A, d_B, N);

    hipDeviceSynchronize();

    // Copy result back
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
}
