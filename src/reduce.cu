#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include "reduce.cuh"

__global__ void reduceKernel(const float* input, float* output, int N) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    float sum = 0.0f;

    if (i < N)
        sum = input[i];
    if (i + blockDim.x < N)
        sum += input[i + blockDim.x];

    sdata[tid] = sum;
    __syncthreads();

    // In-block reduction
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s)
            sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid == 0)
        output[blockIdx.x] = sdata[0];
}

float reduceSum(const float* data, int N) {
    float *dIn, *dOut;
    size_t size = N * sizeof(float);
    hipMalloc(&dIn, size);

    int threads = 256;
    int blocks = (N + threads * 2 - 1) / (threads * 2);
    hipMalloc(&dOut, blocks * sizeof(float));

    hipMemcpy(dIn, data, size, hipMemcpyHostToDevice);

    reduceKernel<<<blocks, threads, threads * sizeof(float)>>>(dIn, dOut, N);

    float *hOut = new float[blocks];
    hipMemcpy(hOut, dOut, blocks * sizeof(float), hipMemcpyDeviceToHost);

    float total = 0.0f;
    for (int i = 0; i < blocks; ++i)
        total += hOut[i];

    delete[] hOut;
    hipFree(dIn);
    hipFree(dOut);

    return total;
}
