#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include "reduce.cuh"

// B = sum(A)
__global__ void reduceKernel(const float* A, float* B, int N) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    float sum = 0.0f;

    if (i < N)
        sum = A[i];
    if (i + blockDim.x < N)
        sum += A[i + blockDim.x];

    sdata[tid] = sum;
    __syncthreads();

    // In-block reduction
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s)
            sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid == 0)
        B[blockIdx.x] = sdata[0];
}

float reduceSum(const float* A, int N) {
    float *dIn, *dOut;
    size_t size = N * sizeof(float);
    hipMalloc(&dIn, size);

    int threads = 256;
    int blocks = (N + threads * 2 - 1) / (threads * 2);
    hipMalloc(&dOut, blocks * sizeof(float));

    hipMemcpy(dIn, A, size, hipMemcpyHostToDevice);

    reduceKernel<<<blocks, threads, threads * sizeof(float)>>>(dIn, dOut, N);

    float *hOut = new float[blocks];
    hipMemcpy(hOut, dOut, blocks * sizeof(float), hipMemcpyDeviceToHost);

    float total = 0.0f;
    for (int i = 0; i < blocks; ++i)
        total += hOut[i];

    delete[] hOut;
    hipFree(dIn);
    hipFree(dOut);

    return total;
}
